#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/softmax_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SoftmaxLossForwardGPU(const int nthreads,
          const Dtype* prob_data, const Dtype* label, Dtype* loss,
          const int num, const int dim, const int spatial_dim,
          const bool has_ignore_label_, const int ignore_label_,
          Dtype* counts) {

   // XXX notice this 
  CUDA_KERNEL_LOOP(index, nthreads) {

     // XXX
     // data_at(0,0,0,0) is the class index
     // data_at(0,1,0,0) is the weight



    const int n = index / spatial_dim;
    const int s = index % spatial_dim;
    const int target = n * spatial_dim + s;
    int j;
    loss[index] = 0;
    for(j = 0; j < 10; j+=1) {
      const int label_value      = static_cast<int>(label[target*10*2 + j*2 + 0]);
      const float weight_value = static_cast<float>(label[target*10*2 + j*2 + 1]);

      if(label_value >= 0 && weight_value > 0) { 
         loss[index] -= weight_value * log(max(prob_data[n * dim + label_value * spatial_dim + s], Dtype(FLT_MIN)));
      }
    }
    counts[index] = 1;
  }
}

template <typename Dtype>
void SoftmaxWithLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  softmax_layer_->Forward(softmax_bottom_vec_, softmax_top_vec_);
  const Dtype* prob_data = prob_.gpu_data();
  const Dtype* label = bottom[1]->gpu_data();
  const Dtype* lcpu = bottom[1]->cpu_data();
  const int dim = prob_.count() / outer_num_;
  const int nthreads = outer_num_ * inner_num_;


  /* int index; */
  /* int spatial_dim = inner_num_; */

  /* for(index=0; index<2; ++index){ */
  /*   const int n = index / spatial_dim; */
  /*   const int s = index % spatial_dim; */
  /*   const int target = n * spatial_dim + s; */
  /*   int j; */

  /*   for(j = 0; j < 10; j+=1) { */
  /*     std::cout << "weight_value " << weight_value << "label_value " << label_value << std::endl; */
  /*   } */
  /* } */


  /* std::cout<< "inn " << inner_num_ << std::endl; */
  /* std::cout<< "out " << outer_num_ << std::endl; */


  /* for(int i=0; i<4; ++i) { */
  /*    std::cout <<"at i: "<<lcpu[i] << std::endl; */
  /* } */

  // Since this memory is not used for anything until it is overwritten
  // on the backward pass, we use it here to avoid having to allocate new GPU
  // memory to accumulate intermediate results in the kernel.
  Dtype* loss_data = bottom[0]->mutable_gpu_diff();
  // Similarly, this memory is never used elsewhere, and thus we can use it
  // to avoid having to allocate additional GPU memory.
  Dtype* counts = prob_.mutable_gpu_diff();
  // NOLINT_NEXT_LINE(whitespace/operators)
  SoftmaxLossForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, prob_data, label, loss_data,
      outer_num_, dim, inner_num_, has_ignore_label_, ignore_label_, counts);
  Dtype loss;
  caffe_gpu_asum(nthreads, loss_data, &loss);
  Dtype valid_count = -1;
  // Only launch another CUDA kernel if we actually need the count of valid
  // outputs.
  if (normalization_ == LossParameter_NormalizationMode_VALID &&
      has_ignore_label_) {
    caffe_gpu_asum(nthreads, counts, &valid_count);
  }
  top[0]->mutable_cpu_data()[0] = loss / get_normalizer(normalization_,
                                                        valid_count);
  if (top.size() == 2) {
    top[1]->ShareData(prob_);
  }
}

template <typename Dtype>
__global__ void SoftmaxLossBackwardGPU(const int nthreads, const Dtype* top,
          const Dtype* label, Dtype* bottom_diff, const int num, const int dim,
          const int spatial_dim, const bool has_ignore_label_,
          const int ignore_label_, Dtype* counts) {

  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / spatial_dim;
    const int s = index % spatial_dim;
    const int target = n * spatial_dim + s;
    int j;

    for (j = 0; j < 10; j+=1) {
      const int label_value      = static_cast<int>(label[target*10*2 + j*2 + 0]);
      const float weight_value = static_cast<float>(label[target*10*2 + j*2 + 1]);

      if (label_value >= 0 && weight_value > 0) { 
         bottom_diff[n * dim + label_value * spatial_dim + s] -= weight_value;
      }
    }
    counts[index] = 1;
  }
}

template <typename Dtype>
void SoftmaxWithLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to label inputs.";
  }
  if (propagate_down[0]) {
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const Dtype* prob_data = prob_.gpu_data();
    const Dtype* top_data = top[0]->gpu_data();
    caffe_gpu_memcpy(prob_.count() * sizeof(Dtype), prob_data, bottom_diff);
    const Dtype* label = bottom[1]->gpu_data();
    const int dim = prob_.count() / outer_num_;
    const int nthreads = outer_num_ * inner_num_;
    // Since this memory is never used for anything else,
    // we use to to avoid allocating new GPU memory.
    Dtype* counts = prob_.mutable_gpu_diff();
    // NOLINT_NEXT_LINE(whitespace/operators)
    SoftmaxLossBackwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
        CAFFE_CUDA_NUM_THREADS>>>(nthreads, top_data, label, bottom_diff,
        outer_num_, dim, inner_num_, has_ignore_label_, ignore_label_, counts);

    Dtype valid_count = -1;
    // Only launch another CUDA kernel if we actually need the count of valid
    // outputs.
    if (normalization_ == LossParameter_NormalizationMode_VALID &&
        has_ignore_label_) {
      caffe_gpu_asum(nthreads, counts, &valid_count);
    }
    const Dtype loss_weight = top[0]->cpu_diff()[0] /
                              get_normalizer(normalization_, valid_count);
    caffe_gpu_scal(prob_.count(), loss_weight , bottom_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SoftmaxWithLossLayer);

}  // namespace caffe
